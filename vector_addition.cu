#include "hip/hip_runtime.h"
//---------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
//---------------------------------------------------------------------------------
static const int WORK_SIZE = 200000000;
static const int BLK_SIZE = 256;

using namespace std;
//---------------------------------------------------------------------------------
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

//---------------------------------------------------------------------------------

__global__ void matrixTransposeShared(const int *A_d, int *T_d, const int rows, const int cols) {
	
	int tx=threadIdx.x; int ty=threadIdx.y;
	int row = blockIdx.y * TILE_SIZE + ty;
	int col = blockIdx.x * TILE_SIZE + tx;
	int rowt = blockIdx.x * TILE_SIZE + ty;
	int colt = blockIdx.y * TILE_SIZE + tx;

	__shared__ int A_ds[TILE_SIZE][TILE_SIZE];
	
    int i = by + threadIdx.y; int j = bx + threadIdx.x;
    int ti = bx + threadIdx.y; int tj = by + threadIdx.x;

    if (row<rows && col<cols)
	A_ds[tx][ty] = A_d[row*cols+col];

    __syncthreads();
    if (colt < cols && rowt<rows)
	T_d[rowt * rows+colt] = A_ds[ty][tx];
}
//---------------------------------------------------------------------------------
int main(void) {
	unsigned int *A_h;
	unsigned int *A_d;
	unsigned int *B_h;
	unsigned int *B_d;
	unsigned int *C_h;
	unsigned int *C_d;

	//Set Device
	CUDA_CHECK_RETURN(hipSetDevice(0));

	//See random number generator
	srand(time(NULL));

	//Clear command prompt
	cout << "\033[2J\033[1;1H";

	cout << "Allocating arrays on host ... ";
	A_h = new unsigned int[WORK_SIZE];
	B_h = new unsigned int[WORK_SIZE];
	C_h = new unsigned int[WORK_SIZE];

	cout << "done.\nPopluating arrays on host ... ";
	for (int i = 0; i < WORK_SIZE; i++) {
		A_h[i] = rand();
		B_h[i] = rand();
	}

	cout << "done.\nAllocating arrays on device ... ";
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &A_d, sizeof(unsigned int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &B_d, sizeof(unsigned int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMalloc((void** ) &C_d, sizeof(unsigned int) * WORK_SIZE));

	cout << "done.\nCopying arrays from host to device ... ";
	CUDA_CHECK_RETURN(
			hipMemcpy(A_d, A_h, sizeof(int) * WORK_SIZE,
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(
			hipMemcpy(B_d, B_h, sizeof(int) * WORK_SIZE,
					hipMemcpyHostToDevice));

	cout << "done.\nLaunching kernel ... ";

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** define kernel launch parameters ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	//Time kernel launch
	//Time kernel launch
	hipEvent_t start, stop;
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
	float elapsedTime;

	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Add kernel call here ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError()); //Check if an error occurred in device code
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed kernel time: " << elapsedTime << " ms\n";

	cout << "Copying results back to host .... ";
	CUDA_CHECK_RETURN(
			hipMemcpy(C_h, C_d, sizeof(int) * WORK_SIZE,
					hipMemcpyDeviceToHost));

	cout << "done.\nVerifying results on host ... ";

	//Add code to time host calculations
	clock_t st, ed;

	st = clock();

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Add validation code here ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	ed = clock() - st;
	
	cout << "done\n";
	
	cout << "Elapsed time on host: " << ((float) ed) / CLOCKS_PER_SEC * 1000
			<< " ms" << endl;	

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// **** Output whether results are valid ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@			

	cout << "Freeing memory on device ... ";
	CUDA_CHECK_RETURN(hipFree((void* ) A_d));
	CUDA_CHECK_RETURN(hipFree((void* ) B_d));
	CUDA_CHECK_RETURN(hipFree((void* ) C_d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	cout << "done.\nFreeing memory on host ... ";
	delete[] A_h;
	delete[] B_h;
	delete[] C_h;

	cout << "done.\nExiting program.\n";
	return 0;
}
