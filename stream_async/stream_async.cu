//------------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <iostream>
//------------------------------------------------------------------------
using namespace std;
//------------------------------------------------------------------------
static const int TILE_SIZE = 512;
static const int SHIFT_VALUE = 10;
static const int NUM_OF_FILES = 6;
static const int MAX_FILE_LENGTH = 60000; 
static const long DEVICE = 0;
__constant__  int  S;
//------------------------------------------------------------------------
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
 //------------------------------------------------------------------------
__global__ void decrypt_caesar_cipher(char* N, char* P, int length) {

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// ****      Write Kernel Code       ****
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

}
//---------------------------------------------------------------------------------------------------
int loadInputFile(string fName, char* inputArray) {
	ifstream inputFile;

	inputFile.open(fName.c_str());
	int cnt = 0;
	if (inputFile.is_open()) {
		char temp;
		while (inputFile.get(temp)) {
			inputArray[cnt++] = temp;
		}
		inputFile.close();
	}
	return cnt;
}
//---------------------------------------------------------------------------------------------------
void writeOutput(string oName, char* output, int size) {
	ofstream outputFile;

	outputFile.open(oName.c_str());
	if (outputFile.is_open()) {
		for (size_t i = 0; i < size; i++) {
			outputFile << output[i];
		}
		outputFile.close();
	}
}
//---------------------------------------------------------------------------------------------------
int main(void) {

	//Clear command prompt
	cout << "\033[2J\033[1;1H";

	//Initalize random number generator
	srand(time(NULL));

	//Set device
	CUDA_CHECK_RETURN(hipSetDevice(DEVICE));

	//Create input array
	cout << "Allocating input array on host ... ";
	int* file_lengths = new int[NUM_OF_FILES];
	char** h_N = new char* [NUM_OF_FILES];
	char** h_P = new char* [NUM_OF_FILES];
	char* d_N;
	char* d_P;
	//Create input array on device
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_N, sizeof(char) * MAX_FILE_LENGTH));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_P, sizeof(char) * MAX_FILE_LENGTH));

	cout << "done.\nLoading input data ... ";
	for (int i = 0; i < NUM_OF_FILES; i++) {
		h_N[i] = new char[MAX_FILE_LENGTH];
		h_P[i] = new char[MAX_FILE_LENGTH];
		int temp = loadInputFile("./encrypted" + to_string(i) + ".txt", h_N[i]);
		file_lengths[i] = temp;
	}

	cout << "done.\nCopying shift to device ... ";
	CUDA_CHECK_RETURN(
		hipMemcpyToSymbol(HIP_SYMBOL(S), &SHIFT_VALUE, sizeof(int)));
	cout << "done." << endl;	

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// *** Define kernel parameters here ***
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	
	cout << "Launching " << NUM_OF_FILES << " kernels on default stream ... ";		

	hipEvent_t start, stop;
	float elapsedTime;
	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
		
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// ** Make kernel calls on default stream here  **
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	

	hipEventRecord(stop, 0);

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed time on default stream: " << elapsedTime << " ms\n";

	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// ***  Define & create streams objects here ***
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@

	cout << "\nLaunching " << NUM_OF_FILES << " kernels with " << NUM_OF_FILES << " streams ... ";

	CUDA_CHECK_RETURN(hipEventCreate(&start));
	CUDA_CHECK_RETURN(hipEventCreate(&stop));
	CUDA_CHECK_RETURN(hipEventRecord(start, 0));
		
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	// ***  Make stream kernel calls here  ***
	//@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
	

    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
    // ***  Free stream objects memory here  ***    
    //@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@@
    
	hipEventRecord(stop, 0);

	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipEventDestroy(start));
	CUDA_CHECK_RETURN(hipEventDestroy(stop));
	cout << "done.\nElapsed time with " << NUM_OF_FILES << " kernels: " << elapsedTime << " ms\n\n";

	cout << "Writing output ... ";
	for (int i = 0; i < NUM_OF_FILES; i++)
		writeOutput("decrypted" + to_string(i) + ".txt", h_P[i], file_lengths[i]);

	cout << "done.\nFreeing memory ...";
	CUDA_CHECK_RETURN(hipFree((void*)d_N));
	CUDA_CHECK_RETURN(hipFree((void*)d_P));
	CUDA_CHECK_RETURN(hipDeviceReset());
	
	for (int i = 0; i < NUM_OF_FILES; i++) {
		delete[] h_N[i];
		delete[] h_P[i];
	}
	delete[] h_P;
	delete[] h_N;
	delete[] file_lengths;

	cout << "done.\nExiting program\n";
	
	return 0;
}
//---------------------------------------------------------------------------------------------------